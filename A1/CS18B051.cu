#ifndef _KERNELS_H_
#define _KERNELS_H_


__global__ void per_row_kernel(int m,int n,int *A,int *B,int *C){
    unsigned long long int id = blockIdx.x * blockDim.x + threadIdx.x ;
    for(int ii=0;ii<n;ii++){
        if(id<m){
            C[id*n+ii] = A[id*n+ii] + B[id*n+ii];
        }
    }
}


__global__ void per_column_kernel(int m,int n,int *A,int *B,int *C){
    unsigned long long int id = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.y + threadIdx.y ;
    for(int jj=0;jj<m;jj++){
        if(id<n){
            C[jj*n+id] = A[jj*n+id] + B[jj*n+id];
        }
    }
}

__global__ void per_element_kernel(int m,int n,int *A,int *B,int *C){
	unsigned long long int id = ( blockIdx.x * gridDim.y + blockIdx.y ) * blockDim.x * blockDim.y + threadIdx.x * blockDim.y + threadIdx.y ;
	if(id < m*n){
		int i = id/n;
		int j = id%n;
		C[i*n+j] = A[i*n+j] + B[i*n+j];
	}
}

#endif